
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <complex>
#include <chrono>
#include <cstdio>
#include <random>

using namespace std;

const int NUM_TILES_1D = 100;
const int SAMPLES_IN_BATCH = 100;
const double width = 3. / NUM_TILES_1D;
const double height = 3. / NUM_TILES_1D;
const double EPSILON = 0.01;

bool AreSame(double a, double b)
{
  return fabs(a - b) < EPSILON;
}

bool is_in_mandelbrot(const double x, const double y)
{
  // Tortoise and Hare approach to check if point (x, y) is in Mandelbrot set.
  std::complex<double> z_hare(0.0, 0.0);
  std::complex<double> z_tortoise(0.0, 0.0);
  std::complex<double> c(x, y);
  while (true)
  {
    z_hare = z_hare * z_hare + c;
    z_hare = z_hare * z_hare + c;
    z_tortoise = z_tortoise * z_tortoise + c;
    if (z_hare == z_tortoise)
    {
      return true;
    }
    float criteria = std::pow(z_hare.real(), 2) + std::pow(z_hare.imag(), 2);
    if (criteria > 4.0)
    {
      return false;
    }
  }
}

double count_mandelbrot(mt19937 rng, int num_samples, double x_min, double width, double y_min, double height)
{
  double out = 0.;

  // Random number generator distribution
  std::uniform_real_distribution<double> dist(0.0f, 1.0f); // Uniform distribution between 0 and 1

  for (int i = 0; i < num_samples; ++i)
  {
    double x_norm = dist(rng);
    double y_norm = dist(rng);
    double x = x_min + (x_norm * width);
    double y = y_min + (y_norm * height);

    if (is_in_mandelbrot(x, y))
    {
      out += 1.;
    }
  }
  return out;
}

// __global__ void hello_world_gpu()
// {
//   printf("Hello World from the GPU at block %u, thread %u \n", blockIdx.x, threadIdx.x);
// }

// void hello_world_cpu()
// {
//   printf("Hello World from the CPU \n");
// }

double wald_uncertainty(double numer, double denom)
{
  if (AreSame(numer, 0.))
  {
    numer = 1.0;
    denom++;
  }
  else if (AreSame(numer, denom))
  {
    denom++;
  }

  double frac = numer / denom;
  return sqrt(frac * (1.0 - frac) / denom);
}

double xmin(int j)
{
  return -2 + width * j;
}

double ymin(int i)
{
  return -3 / 2 + height * i;
}

void compute_until(vector<mt19937> &random_generators, vector<vector<double>> &numer, vector<vector<double>> &denom, vector<vector<double>> &uncert, double uncert_target)
{
  for (int i = 0; i < NUM_TILES_1D; i++)
    for (int j = 0; j < NUM_TILES_1D; j++)
    {
      mt19937 rng = random_generators[NUM_TILES_1D * i + j];
      uncert[i][j] = numeric_limits<double>::infinity();
      while (uncert[i][j] > uncert_target)
      {
        denom[i][j] += (double)SAMPLES_IN_BATCH;
        numer[i][j] += (double)count_mandelbrot(rng, SAMPLES_IN_BATCH, xmin(j), width, ymin(i), height);
        uncert[i][j] = wald_uncertainty(numer[i][j], denom[i][j]) * width * height;
      }
    }
}

int main(int argc, char *argv[])
{
  vector<vector<double>> numer(NUM_TILES_1D, vector<double>(NUM_TILES_1D));
  vector<vector<double>> denom(NUM_TILES_1D, vector<double>(NUM_TILES_1D));
  vector<vector<double>> uncert(NUM_TILES_1D, vector<double>(NUM_TILES_1D));

  for (int i = 0; i < NUM_TILES_1D; i++)
    for (int j = 0; j < NUM_TILES_1D; j++)
    {
      numer[i][j] = 0.;
      denom[i][j] = 0.;
      uncert[i][j] = 0.;
    }

  vector<mt19937> rngs(NUM_TILES_1D * NUM_TILES_1D);
  for (int i = 0; i < NUM_TILES_1D * NUM_TILES_1D; i++)
    rngs[i].seed(i);

  compute_until(rngs, numer, denom, uncert, 1e-3);
  double final_value = 0;
  for (int i = 0; i < NUM_TILES_1D; i++)
    for (int j = 0; j < NUM_TILES_1D; j++)
    {
      final_value += (numer[i][j] / denom[i][j]) * width * height;
    }

  // // hello_world_cpu();

  // // const int n_blocks = atoi(argv[argc - 2]);
  // // const int n_threads = atoi(argv[argc - 1]);

  // // dim3 grid_dim(n_blocks);
  // // dim3 block_dim(n_threads);

  // // hello_world_gpu<<<grid_dim, block_dim>>>();

  // // cudaDeviceSynchronize();

  return 0;
}
