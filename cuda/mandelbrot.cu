
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>

using namespace std;

__global__ void hello_world_gpu()
{
  printf("Hello World from the GPU at block %u, thread %u \n", blockIdx.x, threadIdx.x);
}

void hello_world_cpu()
{
  printf("Hello World from the CPU \n");
}

double wald_uncertainty(double numer, double denom)
{
  if (numer == 0)
  {
    numer = 1.0;
    denom++;
  }
  else if (numer == denom)
    denom++;

  double frac = numer / denom;
  return sqrt(frac * (1.0 - frac) / denom);
}

int main(int argc, char *argv[])
{

  if (argc != 3)
  {
    cout << "Need two arguments: number of blocks and number of threads" << endl;
    return -1;
  }

  hello_world_cpu();

  const int n_blocks = atoi(argv[argc - 2]);
  const int n_threads = atoi(argv[argc - 1]);

  dim3 grid_dim(n_blocks);
  dim3 block_dim(n_threads);

  hello_world_gpu<<<grid_dim, block_dim>>>();

  hipDeviceSynchronize();

  return 0;
}
